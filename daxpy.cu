#include <hip/hip_runtime.h>

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

clock_t diff, start, end;

__global__ void daxpy(const int n, double *x, double *y, double *z){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if(i < n){
    z[i] = 35.0*x[i] + y[i];
  }
}

int main(void){
  hipError_t err = hipSuccess;
  
  unsigned numElements = 50000;

  size_t size = numElements * sizeof(double);
  printf("addition of %d elements]\n",numElements);

  // Allocate the host input vector A
  double *h_A = (double *)malloc(size);

  // Allocate the host input vector B
  double *h_B = (double *)malloc(size);

  // Allocate the host input vector B
  double *h_C = (double *)malloc(size);

  // Verify that allocations succeeded
  if (h_A == NULL || h_B == NULL || h_C == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    exit(EXIT_FAILURE);
  }

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) {
    h_A[i] = rand() / (double)RAND_MAX;
    h_B[i] = rand() / (double)RAND_MAX;
    h_C[i] = rand() / (double)RAND_MAX;
  }

  double *d_A = NULL;
  err = hipMalloc((void **)&d_A, size);

  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }

  double *d_B = NULL;
  err = hipMalloc((void **)&d_B, size);

  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }

  double *d_C = NULL;
  err = hipMalloc((void **)&d_C, size);

  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }


  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }

  int threadsPerBlock = 256;

  // TODO : understand why numElements + threadsPerBlock?
  int blocksPerGrid = (numElements + threadsPerBlock -1) / threadsPerBlock;

  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);

   start = clock();
  daxpy<<<blocksPerGrid, threadsPerBlock>>>(numElements, d_A, d_B, d_C);
  end = clock();
  
  err = hipGetLastError();
  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }

    // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  printf("Time Taken : %ld \n", end-start);
  // Verify that the result vector is correct
  for (int i = 0; i < numElements; ++i) {
    if (fabs(35.0*h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Test PASSED\n");

  // Free device global memory
  err = hipFree(d_A);
  
  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_B);

  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_C);
  
  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }
  
  
  free(h_A);
  free(h_B);
  return 0;
}
